/* This module setup work around CUDA different devices, allowing for better
fitting to the actual GPU device
*/
#ifndef DICOMNODE_CUDA_MANAGEMENT
#define DICOMNODE_CUDA_MANAGEMENT

#include<stdint.h>

#include<pybind11/pybind11.h>

namespace py = pybind11;

hipDeviceProp_t get_current_device(){
  hipDeviceProp_t prop;
  int current_device;
  hipGetDevice(&current_device);
  hipGetDeviceProperties(&prop, current_device);
  return prop;
}

py::object cast_current_device(){
  return py::cast(get_current_device());
}

template<class R, class... Args>
int32_t maximize_shared_memory(R (func)(Args...)){
  hipDeviceProp_t dev_prop = get_current_device();
  if (dev_prop.major >= 7){
    hipFuncSetAttribute(reinterpret_cast<const void*>(func), hipFuncAttributeMaxDynamicSharedMemorySize, dev_prop.sharedMemPerBlockOptin);
  }
  return dev_prop.sharedMemPerBlockOptin;
}


void apply_cuda_management_module(py::module& m){
  py::class_<hipDeviceProp_t>(m, "DeviceProperties")
    .def_readonly("major", &hipDeviceProp_t::major)
    .def_readonly("minor", &hipDeviceProp_t::minor)
    .def_readonly("totalGlobalMem", &hipDeviceProp_t::totalGlobalMem)
    .def_readonly("totalConstMem", &hipDeviceProp_t::totalConstMem)
    .def_readonly("name", &hipDeviceProp_t::name)
    .def_readonly("mangedMemory", &hipDeviceProp_t::managedMemory)
    .def_readonly("sharedMemPerBlock", &hipDeviceProp_t::sharedMemPerBlock)
    .def_readonly("sharedMemPerBlockOptin", &hipDeviceProp_t::sharedMemPerBlockOptin)
    .def_readonly("sharedMemPerMultiprocessor", &hipDeviceProp_t::sharedMemPerMultiprocessor)
    .def_readonly("unifiedAddressing", &hipDeviceProp_t::unifiedAddressing)
    .def_readonly("unifiedFunctionPointers", &hipDeviceProp_t::unifiedFunctionPointers)
    .def_readonly("concurrentKernels", &hipDeviceProp_t::concurrentKernels)
    .def_readonly("concurrentManagedAccess", &hipDeviceProp_t::concurrentManagedAccess)
    .def_readonly("directManagedMemAccessFromHost", &hipDeviceProp_t::directManagedMemAccessFromHost);

  m.def("get_device_properties", &cast_current_device);
}


#endif