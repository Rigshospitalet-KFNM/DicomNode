
#include <hip/hip_runtime.h>
/* This module setup work around CUDA different devices, allowing for better
fitting to the actual GPU device

*/

hipDeviceProp_t get_current_device(){
  hipDeviceProp_t prop;
  int current_device;
  hipGetDevice(&current_device);
  hipGetDeviceProperties(&prop, current_device);
  return prop;
}