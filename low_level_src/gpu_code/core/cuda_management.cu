#pragma once

/* This module setup work around CUDA different devices, allowing for better
fitting to the actual GPU device
*/
// C includes
#include<stdint.h>
// STL includes
#include<iostream>
#include<string>
#include<functional>
// Pybind includes
#include<pybind11/pybind11.h>
namespace py = pybind11;

#if defined(__HIPCC__) // NVCC
   #define ALIGN(n) __align__(n)
#elif defined(__GNUC__) // GCC
  #define ALIGN(n) __attribute__((aligned(n)))
#elif defined(_MSC_VER) // MSVC
  #define ALIGN(n) __declspec(align(n))
#else
  #error "Please provide a definition for ALIGN macro for your host compiler!"
#endif

template<typename... Ts>
void free_device_memory(Ts** && ... device_pointer){
  ([&]{
    hipPointerAttribute_t attr;
    hipError_t error = hipPointerGetAttributes(&attr, *device_pointer);
    if(error != hipSuccess){
      std::cout << "something went wrong!\n";
      return;
    }
    if(attr.type == hipMemoryType::hipMemoryTypeDevice || attr.type == hipMemoryType::hipMemoryTypeManaged){
      error = hipFree(*device_pointer);
      if(error != hipSuccess){
        std::cout << "freeing failed!";
      }
      *device_pointer = nullptr;
    }
  }(), ...);
}

class CudaRunner {
  std::function<void(hipError_t)> error_function;
  hipError_t m_error = hipSuccess;
  public:
    hipError_t error() const {
      return m_error;
    }
    CudaRunner(std::function<void(hipError_t)> error_lambda) : error_function(error_lambda){}
    CudaRunner& operator|(std::function<hipError_t()> func){
       if(m_error == hipSuccess){
        m_error = func();
        if (m_error != hipSuccess){
          error_function(m_error);
        }
      }
      return *this;
    };
};

void run_cuda(std::function<hipError_t()> action_function,
              std::function<void(hipError_t)> error_function){
    hipError_t error = action_function();
    if(error != hipSuccess){
        error_function(error);
    }
}


hipDeviceProp_t get_current_device(){
  hipDeviceProp_t prop;
  int current_device;
  hipGetDevice(&current_device);
  hipGetDeviceProperties(&prop, current_device);
  return prop;
}

py::object cast_current_device(){
  return py::cast(get_current_device());
}

template<class R, class... Args>
int32_t maximize_shared_memory(R (kernel)(Args...)){
  hipDeviceProp_t dev_prop = get_current_device();
  if (dev_prop.major >= 7){
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, dev_prop.sharedMemPerBlockOptin);
  }
  return dev_prop.sharedMemPerBlockOptin;
}

void apply_cuda_management_module(py::module& m){
  py::class_<hipDeviceProp_t>(m, "DeviceProperties")
    .def_readonly("major", &hipDeviceProp_t::major)
    .def_readonly("minor", &hipDeviceProp_t::minor)
    .def_readonly("totalGlobalMem", &hipDeviceProp_t::totalGlobalMem)
    .def_readonly("totalConstMem", &hipDeviceProp_t::totalConstMem)
    .def_readonly("name", &hipDeviceProp_t::name)
    .def_readonly("mangedMemory", &hipDeviceProp_t::managedMemory)
    .def_readonly("sharedMemPerBlock", &hipDeviceProp_t::sharedMemPerBlock)
    .def_readonly("sharedMemPerBlockOptin", &hipDeviceProp_t::sharedMemPerBlockOptin)
    .def_readonly("sharedMemPerMultiprocessor", &hipDeviceProp_t::sharedMemPerMultiprocessor)
    .def_readonly("unifiedAddressing", &hipDeviceProp_t::unifiedAddressing)
    .def_readonly("unifiedFunctionPointers", &hipDeviceProp_t::unifiedFunctionPointers)
    .def_readonly("concurrentKernels", &hipDeviceProp_t::concurrentKernels)
    .def_readonly("concurrentManagedAccess", &hipDeviceProp_t::concurrentManagedAccess)
    .def_readonly("directManagedMemAccessFromHost", &hipDeviceProp_t::directManagedMemAccessFromHost);

  m.def("get_device_properties", &cast_current_device);
}
