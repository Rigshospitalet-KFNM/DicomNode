#pragma once

/* This module setup work around CUDA different devices, allowing for better
fitting to the actual GPU device
*/


//#include"cuda_management.cuh"
#include<iostream>
#include<stdint.h>

#include<pybind11/pybind11.h>

namespace py = pybind11;

inline std::ostream& operator<<(std::ostream& os, const hipMemoryType cmt){
    switch(cmt){
      case cudaMemoryTypeUnregistered:
        os << "cudaMemoryTypeUnregistered";
        break;
      case hipMemoryTypeHost:
        os << "hipMemoryTypeHost";
        break;
      case hipMemoryTypeDevice:
        os << "hipMemoryTypeDevice";
        break;
      case hipMemoryTypeManaged:
        os << "hipMemoryTypeManaged";
        break;
  }
  return os;
}

template<typename T>
hipError_t get_pointer_type(const T* const pointer,
                             hipMemoryType& pointer_type){
  hipPointerAttribute_t attributes;
  hipError_t error = hipPointerGetAttributes(&attributes, pointer);
  pointer_type = attributes.type;

  return error;
}

template<typename... Ts>
void free_device_memory(Ts** && ... device_pointer){
  ([&]{
    hipPointerAttribute_t attr;
    hipError_t error = hipPointerGetAttributes(&attr, *device_pointer);
    if(error != hipSuccess){
      std::cout << "something went wrong!\n";
      return;
    }
    if(attr.type == hipMemoryType::hipMemoryTypeDevice || attr.type == hipMemoryType::hipMemoryTypeManaged){
      error = hipFree(*device_pointer);
      if(error != hipSuccess){
        std::cout << "freeing failed!";
      }
      *device_pointer = nullptr;
    }
  }(), ...);
}

class CudaRunner {
  std::function<void(hipError_t)> error_function;
  public:
    hipError_t error = hipSuccess;
    CudaRunner(std::function<void(hipError_t)> error_lambda) : error_function(error_lambda){}
    CudaRunner& operator|(std::function<hipError_t()> func){
       if(error == hipSuccess){
        error = func();
        if (error != hipSuccess){
          error_function(error);
        }
      }
      return *this;
    };
};

void run_cuda(std::function<hipError_t()> action_function,
              std::function<void(hipError_t)> error_function){
    hipError_t error = action_function();
    if(error != hipSuccess){
        error_function(error);
    }
}


hipDeviceProp_t get_current_device(){
  hipDeviceProp_t prop;
  int current_device;
  hipGetDevice(&current_device);
  hipGetDeviceProperties(&prop, current_device);
  return prop;
}

py::object cast_current_device(){
  return py::cast(get_current_device());
}

template<class R, class... Args>
int32_t maximize_shared_memory(R (kernel)(Args...)){
  hipDeviceProp_t dev_prop = get_current_device();
  if (dev_prop.major >= 7){
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, dev_prop.sharedMemPerBlockOptin);
  }
  return dev_prop.sharedMemPerBlockOptin;
}


void apply_cuda_management_module(py::module& m){
  py::class_<hipDeviceProp_t>(m, "DeviceProperties")
    .def_readonly("major", &hipDeviceProp_t::major)
    .def_readonly("minor", &hipDeviceProp_t::minor)
    .def_readonly("totalGlobalMem", &hipDeviceProp_t::totalGlobalMem)
    .def_readonly("totalConstMem", &hipDeviceProp_t::totalConstMem)
    .def_readonly("name", &hipDeviceProp_t::name)
    .def_readonly("mangedMemory", &hipDeviceProp_t::managedMemory)
    .def_readonly("sharedMemPerBlock", &hipDeviceProp_t::sharedMemPerBlock)
    .def_readonly("sharedMemPerBlockOptin", &hipDeviceProp_t::sharedMemPerBlockOptin)
    .def_readonly("sharedMemPerMultiprocessor", &hipDeviceProp_t::sharedMemPerMultiprocessor)
    .def_readonly("unifiedAddressing", &hipDeviceProp_t::unifiedAddressing)
    .def_readonly("unifiedFunctionPointers", &hipDeviceProp_t::unifiedFunctionPointers)
    .def_readonly("concurrentKernels", &hipDeviceProp_t::concurrentKernels)
    .def_readonly("concurrentManagedAccess", &hipDeviceProp_t::concurrentManagedAccess)
    .def_readonly("directManagedMemAccessFromHost", &hipDeviceProp_t::directManagedMemAccessFromHost);

  m.def("get_device_properties", &cast_current_device);
}
