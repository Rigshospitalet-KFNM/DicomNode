#include <gtest/gtest.h>
#include"../gpu_code/dicom_node_gpu.cuh"

#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>

#include<iostream>

TEST(CUDA_MANAGEMENT_TESTS, Free_Device_memory) {
  int* ptr;
  hipError_t error = hipMalloc(&ptr, sizeof(int) * 10);
  EXPECT_EQ(error, hipSuccess);
  free_device_memory(&ptr);
  EXPECT_EQ(ptr, nullptr);
}

TEST(CUDA_MANAGEMENT_TESTS, CUDA_RUNNER_TEST){
  int function_flag = 0;
  hipError_t error_flag = hipSuccess;

  auto error_function = [&](hipError_t input){
    error_flag = input;
  };

  CudaRunner runner{error_function};
  runner | [&](){
      function_flag = 1;
      return hipSuccess;
    } | [](){
      return hipErrorNotInitialized;
    } | [&](){
      function_flag = 2;
      return hipSuccess;
    };

  EXPECT_TRUE(function_flag == 1);
  EXPECT_TRUE(error_flag == hipErrorNotInitialized);
  EXPECT_TRUE(runner.error() == hipErrorNotInitialized);
}
