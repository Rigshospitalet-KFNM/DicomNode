#include <gtest/gtest.h>

#include"../gpu_code/dicom_node_gpu.cu"

#include<iostream>

TEST(CUDA_MANAGEMENT_TESTS, Free_Device_memory) {
  int* ptr;
  hipError_t error = hipMalloc(&ptr, sizeof(int) * 10);
  ASSERT_EQ(error, hipSuccess);
  free_device_memory(&ptr);
  ASSERT_EQ(ptr, nullptr);
}

TEST(CUDA_MANAGEMENT_TESTS, CUDA_RUNNER_TEST){
  int function_flag = 0;
  hipError_t error_flag = hipSuccess;

  auto error_function = [&](hipError_t input){
    error_flag = input;
  };

  CudaRunner runner{error_function};
  runner | [&](){
      function_flag = 1;
      return hipSuccess;
    } | [](){
      return hipErrorNotInitialized;
    } | [&](){
      function_flag = 2;
      return hipSuccess;
    };

  std::cout << (function_flag == 1) << "\n";
  std::cout << (error_flag == hipErrorNotInitialized) << "\n";
  std::cout << (runner.error() == hipErrorNotInitialized) << "\n";
}
