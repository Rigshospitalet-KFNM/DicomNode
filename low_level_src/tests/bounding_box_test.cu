#pragma once

#include<stdint.h>

#include <gtest/gtest.h>
#include"../gpu_code/dicom_node_gpu.cu"

TEST(BOUNDING_BOX, Box_25x25x25_with_10_11_12_20_19_18){
  hipError_t status;
  const size_t elements = 25 * 25 * 25;

  auto idx = [](int i, int j, int k) constexpr {
    return k * 25 * 25 + j * 25 + i;
  };

  const size_t datasize = elements * sizeof(uint8_t);
  uint8_t* host_data = (uint8_t*)malloc(datasize);
  ASSERT_NE(host_data, (void*)NULL);
  memset(host_data, (uint8_t)0, datasize);
  host_data[idx(10,11,12)] = 1;
  host_data[idx(20,19,18)] = 1;
  BoundingBox_3D out;

  status = reduce<1, BoundingBoxOP_3D<uint8_t>, uint8_t, BoundingBox_3D, Domain<3>>(
    host_data, datasize, &out, {25,25,25}
  );

  ASSERT_EQ(status, hipSuccess);
  ASSERT_EQ(out.x_min, 10);
  ASSERT_EQ(out.x_max, 20);
  ASSERT_EQ(out.y_min, 11);
  ASSERT_EQ(out.y_max, 19);
  ASSERT_EQ(out.z_min, 12);
  ASSERT_EQ(out.z_max, 18);

  free(host_data);
}
