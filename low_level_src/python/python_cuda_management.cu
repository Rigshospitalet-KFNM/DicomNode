// Pybind includes
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>

#include"utilities.cuh"

#include"../gpu_code/dicom_node_gpu.cu"


pybind11::object cast_current_device(){
  return pybind11::cast(get_current_device());
}

void apply_cuda_management_module(pybind11::module& m){
  pybind11::class_<hipDeviceProp_t>(m, "DeviceProperties")
    .def_readonly("major", &hipDeviceProp_t::major)
    .def_readonly("minor", &hipDeviceProp_t::minor)
    .def_readonly("totalGlobalMem", &hipDeviceProp_t::totalGlobalMem)
    .def_readonly("totalConstMem", &hipDeviceProp_t::totalConstMem)
    .def_readonly("name", &hipDeviceProp_t::name)
    .def_readonly("mangedMemory", &hipDeviceProp_t::managedMemory)
    .def_readonly("sharedMemPerBlock", &hipDeviceProp_t::sharedMemPerBlock)
    .def_readonly("sharedMemPerBlockOptin", &hipDeviceProp_t::sharedMemPerBlockOptin)
    .def_readonly("sharedMemPerMultiprocessor", &hipDeviceProp_t::sharedMemPerMultiprocessor)
    .def_readonly("unifiedAddressing", &hipDeviceProp_t::unifiedAddressing)
    .def_readonly("unifiedFunctionPointers", &hipDeviceProp_t::unifiedFunctionPointers)
    .def_readonly("concurrentKernels", &hipDeviceProp_t::concurrentKernels)
    .def_readonly("concurrentManagedAccess", &hipDeviceProp_t::concurrentManagedAccess)
    .def_readonly("directManagedMemAccessFromHost", &hipDeviceProp_t::directManagedMemAccessFromHost)
    .def("__repr__",
    [](const hipDeviceProp_t& prop){
      std::stringstream ss;
      ss << "-----Cuda Device Properties-----\n"
         << "Name: " << prop.name << "\n"
         << "Compute capability: " << prop.major << "." << prop.minor << "\n"
         << "Total memory: " << get_byte_string(prop.totalGlobalMem) << "\n"
         << "Shared memory: " << get_byte_string(prop.sharedMemPerBlock) << "\n"
         << "Shared memory (optin): " << get_byte_string(prop.sharedMemPerBlock) << "\n"
         << "Registers per block: " << get_byte_string(prop.regsPerBlock) << "\n"
         << "Registers per multiprocessor: " << get_byte_string(prop.regsPerMultiprocessor);
      return ss.str();
    });

  m.def("get_device_properties", &cast_current_device);
}
