#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>

#include<iostream>
#include<string>
#include<tuple>

#include"../gpu_code/dicom_node_gpu.cuh"
#include"utilities.cuh"

using basis_t = pybind11::array_t<float>;
using domain_array = pybind11::array_t<uint32_t>;

template<typename T>
std::tuple<dicomNodeError_t, python_array<T>> interpolate_linear_templated(
  const pybind11::object& image,
  const pybind11::object& new_space
){
  Space<3> destination_space;
  dicomNodeError_t error = load_space(&destination_space, new_space);
  if (error){
    return {error , python_array<T>(1)};
  }
  Space<3> image_space;
  const size_t image_size = get_image_size<T>(image);
  const size_t out_image_size = get_image_size<T>(new_space);

  if(!out_image_size){
    return {dicomNodeError_t::INPUT_TYPE_ERROR, python_array<T>(1)};
  }
  const size_t out_image_elements = out_image_size / sizeof(T);

  const size_t shape[3] = {
    destination_space.domain[0],
    destination_space.domain[1],
    destination_space.domain[2]
  };
  const size_t strides[3] = {
    destination_space.domain[1] * destination_space.domain[2] * sizeof(T),
    destination_space.domain[2] * sizeof(T),
    sizeof(T)
  };

  pybind11::array_t<T> out_array(shape, strides);
  pybind11::buffer_info out_buffer = out_array.request(true);

  T* image_data = nullptr;
  Texture *device_texture = nullptr;
  T* device_out_image = nullptr;

  auto error_function = [&](dicomNodeError_t _){
    free_texture(&device_texture);
    free_device_memory(&device_out_image);
  };

  DicomNodeRunner runner{error_function};
  runner
    | [&](){ return check_buffer_pointers(std::cref(out_buffer), out_image_elements);}
    | [&](){ return get_image_pointer<T>(image, &image_data);}
    | [&](){ return hipMalloc(&device_texture, sizeof(Texture)); }
    | [&](){
      return load_texture<T>(
        device_texture,
        image_data,
        std::cref(destination_space)
      );
    }
    | [&](){ return hipMalloc(&device_out_image, image_size);}
    | [&](){ return gpu_interpolation_linear<T>(
      device_texture,
      std::cref(destination_space),
      device_out_image
    );}
    | [&](){
      return hipMemcpy(out_buffer.ptr, device_out_image, out_image_size, hipMemcpyDefault);
    }
    | [&](){
      free_texture(&device_texture);
      free_device_memory(&device_out_image);
      return dicomNodeError_t::SUCCESS;
    };

  return {runner.error(), out_array};
}

std::tuple<dicomNodeError_t, pybind11::array> interpolate_linear(const pybind11::object& image,
                                   const pybind11::object& new_space
  ){
  const pybind11::array& raw_image = image.attr("raw");
  const std::string dtype = pybind11::str(raw_image.attr("dtype"));

  //Switch statement doesn't work because I am comparing strings
  if(dtype == "float32"){
    return interpolate_linear_templated<float>(image, new_space);
  } else if (dtype == "float64"){
    return interpolate_linear_templated<double>(image, new_space);
  } else if (dtype == "int8"){
    return interpolate_linear_templated<int8_t>(image, new_space);
  } else if (dtype == "int16"){
    return interpolate_linear_templated<int16_t>(image, new_space);
  } else if (dtype == "int32"){
    return interpolate_linear_templated<int32_t>(image, new_space);
  } else if (dtype == "int64"){
    return interpolate_linear_templated<int64_t>(image, new_space);
  } else if (dtype == "uint8"){
    return interpolate_linear_templated<uint8_t>(image, new_space);
  } else if (dtype == "uint16"){
    return interpolate_linear_templated<uint16_t>(image, new_space);
  } else if (dtype == "uint32"){
    return interpolate_linear_templated<uint32_t>(image, new_space);
  } else if (dtype == "uint64"){
    return interpolate_linear_templated<uint64_t>(image, new_space);
  } else if (dtype == "bool") {
    return interpolate_linear_templated<bool>(image, new_space);
  }

  const std::string error_message = "Unsupported dtype:" + dtype;
  throw std::runtime_error(error_message);
}


void apply_interpolation_module(pybind11::module& m){
  pybind11::module sub_module = m.def_submodule(
    "interpolation",
    "This module contains functions for resampling and interpolation.\n"
  );

  sub_module.def("linear", &interpolate_linear);
}