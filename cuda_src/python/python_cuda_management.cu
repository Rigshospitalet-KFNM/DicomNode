// Pybind includes
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>

#include<tuple>

#include"utilities.cuh"

#include"../gpu_code/dicom_node_gpu.cuh"


pybind11::object cast_current_device(){
  return pybind11::cast(get_current_device());
}

void print_image(const pybind11::object& python_image){
  Image<3, float> host_image;
  dicomNodeError_t error = load_image(&host_image, python_image);

  if(error){
    std::cout << "Encoutered dicomnode Error:" << error << "\n";
  } else {
    std::cout << "Starting Point: (" << host_image.space.starting_point[0] << ", "
                                     << host_image.space.starting_point[1] << ", "
                                     << host_image.space.starting_point[2] << ")\n";
    std::cout << "Extent: (" << host_image.space.extent[0] << ", "
                             << host_image.space.extent[1] << ", "
                             << host_image.space.extent[2] << ")\n";
  }

  free_image(&host_image);
}
void apply_cuda_management_module(pybind11::module& m){
  pybind11::class_<hipDeviceProp_t>(m, "DeviceProperties")
    .def_readonly("major", &hipDeviceProp_t::major)
    .def_readonly("minor", &hipDeviceProp_t::minor)
    .def_readonly("totalGlobalMem", &hipDeviceProp_t::totalGlobalMem)
    .def_readonly("totalConstMem", &hipDeviceProp_t::totalConstMem)
    .def_readonly("name", &hipDeviceProp_t::name)
    .def_readonly("multiProcessorCount", &hipDeviceProp_t::multiProcessorCount)
    .def_readonly("maxThreadsPerMultiProcessor", &hipDeviceProp_t::maxThreadsPerMultiProcessor)
    .def_readonly("mangedMemory", &hipDeviceProp_t::managedMemory)
    .def_readonly("sharedMemPerBlock", &hipDeviceProp_t::sharedMemPerBlock)
    .def_readonly("sharedMemPerBlockOptin", &hipDeviceProp_t::sharedMemPerBlockOptin)
    .def_readonly("sharedMemPerMultiprocessor", &hipDeviceProp_t::sharedMemPerMultiprocessor)
    .def_readonly("unifiedAddressing", &hipDeviceProp_t::unifiedAddressing)
    .def_readonly("unifiedFunctionPointers", &hipDeviceProp_t::unifiedFunctionPointers)
    .def_readonly("concurrentKernels", &hipDeviceProp_t::concurrentKernels)
    .def_readonly("concurrentManagedAccess", &hipDeviceProp_t::concurrentManagedAccess)
    .def_readonly("directManagedMemAccessFromHost", &hipDeviceProp_t::directManagedMemAccessFromHost)
    .def_property_readonly("maxTexture3D", [](const hipDeviceProp_t& prop){
      return pybind11::make_tuple(
        prop.maxTexture3D[0],
        prop.maxTexture3D[1],
        prop.maxTexture3D[2]
      );
    })
    .def("__repr__",
    [](const hipDeviceProp_t& prop){
      std::stringstream ss;
      ss << "-----Cuda Device Properties-----\n"
         << "Name: " << prop.name << "\n"
         << "Compute capability: " << prop.major << "." << prop.minor << "\n"
         << "Total memory: " << get_byte_string(prop.totalGlobalMem) << "\n"
         << "Shared memory: " << get_byte_string(prop.sharedMemPerBlock) << "\n"
         << "Shared memory (optin): " << get_byte_string(prop.sharedMemPerBlock) << "\n"
         << "Registers per block: " << get_byte_string(prop.regsPerBlock) << "\n"
         << "Registers per multiprocessor: " << get_byte_string(prop.regsPerMultiprocessor);
      return ss.str();
    });

  pybind11::class_<hipError_t>(m, "CudaError")
    .def("__int__", [](const hipError_t& error){
      return static_cast<int>(error);
    })
    .def("__bool__", [](const hipError_t& error){
      return error != hipSuccess;
    })
    .def("__repr__", [](const hipError_t& error){
      std::stringstream ss;
      ss << hipGetErrorName(error) << " - " << hipGetErrorString(error);

      return ss.str();
    });

  pybind11::class_<dicomNodeError_t>(m, "DicomnodeError")
    .def("__repr__", [](const dicomNodeError_t& error){
      if(!error){
        return std::string("Success");
      }
      if(is_cuda_error(error)) {
        std::stringstream ss;
        hipError_t cuda_error = extract_cuda_error(error);
        ss << "Encoutered cuda error:" << hipGetErrorName(cuda_error) << " - " << hipGetErrorString(cuda_error);
        return ss.str();
      }

      return std::string("ERROR, raise this as a value!");
    })
    .def("__int__", [](const dicomNodeError_t& error){
      return static_cast<uint32_t>(error);
    })
    .def("__bool__",[](const dicomNodeError_t& error){
      return error != dicomNodeError_t::SUCCESS;
    });

  m.def("print_image", &print_image);
  m.def("get_device_properties", &cast_current_device);
}
